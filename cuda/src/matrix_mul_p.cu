#include "hip/hip_runtime.h"
#include <fstream>

#include "matrix_mul.cuh"
#include "utils.cuh"

using namespace phantom::util;
using namespace phantom::arith;
using namespace nexus;

__global__ void negacyclic_shift_poly_coeffmod_kernel(
    const uint64_t *d_poly, size_t poly_degree, size_t shift, DModulus *modulus, size_t coeff_mod_size, uint64_t *d_result) {
  for (size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
       tid < poly_degree * coeff_mod_size;
       tid += blockDim.x * gridDim.x) {
    size_t twr = tid / poly_degree;
    DModulus mod = modulus[twr];

    uint64_t index_raw = shift + tid;
    uint64_t coeff_count_mod_mask = static_cast<uint64_t>(poly_degree) - 1;
    uint64_t index = index_raw & coeff_count_mod_mask;

    if (!(index_raw & static_cast<uint64_t>(poly_degree)) || !*(d_poly + tid)) {
      *(d_result + index) = *(d_poly + tid);
    } else {
      *(d_result + index) = mod.value() - *(d_poly + tid);
    }
  }
}

__global__ void expand_encode_kernel(const double *d_val, size_t poly_modulus_degree, DModulus *modulus, uint64_t *d_p) {
  size_t i = blockIdx.x * blockDim.x + threadIdx.x;

  if (i < poly_modulus_degree) {
    auto coeffd = round(d_val[i] * 10000000000);
    bool is_negative = signbit(coeffd);
    auto coeffu = static_cast<uint64_t>(fabs(coeffd));

    if (is_negative) {
      for (size_t j = 0; j < 2; j++) {
        d_p[i + (j * poly_modulus_degree)] = negate_uint64_mod(
            barrett_reduce_uint64_uint64(coeffu, modulus[j].value(), modulus[j].const_ratio()[1]), modulus[j].value());
      }
    } else {
      for (size_t j = 0; j < 2; j++) {
        d_p[i + (j * poly_modulus_degree)] = barrett_reduce_uint64_uint64(coeffu, modulus[j].value(), modulus[j].const_ratio()[1]);
      }
    }
  }
}

void MMEvaluator::multiply_power_of_x(PhantomCiphertext &encrypted, PhantomCiphertext &destination, int index) {
  auto context = ckks->context;
  auto coeff_count = ckks->degree;
  // auto moduli_gpu = context->gpu_rns_tables().modulus();
  auto param = context->get_context_data(encrypted.params_id()).parms();
  auto moduli = param.coeff_modulus();
  auto coeff_mod_count = param.coeff_modulus().size();
  auto encrypted_count = encrypted.size();
  auto rns_coeff_count = coeff_count * coeff_mod_count;

  const auto &stream = phantom::util::global_variables::default_stream->get_stream();
  // uint64_t gridDimGlb = coeff_count * coeff_mod_count / blockDimGlb.x;

  ckks->evaluator.transform_from_ntt_inplace(encrypted);

  destination = encrypted;

  auto dest_data = new uint64_t[rns_coeff_count * encrypted_count];
  auto ct_data = new uint64_t[rns_coeff_count * encrypted_count];
  hipMemcpy(ct_data, encrypted.data(), rns_coeff_count * encrypted_count * sizeof(uint64_t), hipMemcpyDeviceToHost);

  if (print_count == 0) {
    for (int i = 0; i < 10; i++) {
      cout << ct_data[i] << " ";
    }
    cout << endl;
  }

  for (int i = 0; i < encrypted_count; i++) {
    // negacyclic_shift_poly_coeffmod_kernel<<<gridDimGlb, blockDimGlb, 0, stream>>>(
    //     encrypted.data(i),
    //     coeff_count,
    //     index,
    //     moduli,
    //     coeff_mod_count,
    //     destination.data(i));
    // -----------------------------------------------------------------------------------
    for (int j = 0; j < coeff_mod_count; j++) {
      uint64_t *poly = ct_data + i * rns_coeff_count + j * coeff_count;
      uint64_t *result = dest_data + i * rns_coeff_count + j * coeff_count;

      uint64_t index_raw = index;
      uint64_t coeff_count_mod_mask = static_cast<uint64_t>(coeff_count) - 1;
      for (size_t k = 0; k < coeff_count; k++, poly++, index_raw++) {
        uint64_t index = index_raw & coeff_count_mod_mask;
        if (!(index_raw & static_cast<uint64_t>(coeff_count)) || !*poly) {
          result[index] = *poly;
        } else {
          result[index] = moduli[j].value() - *poly;
        }
      }
    }
  }

  if (print_count == 0) {
    for (int i = 0; i < 10; i++) {
      cout << dest_data[i] << " ";
    }
    cout << endl;
    print_count += 1;
  }

  hipMemcpy(destination.data(), dest_data, rns_coeff_count * encrypted_count * sizeof(uint64_t), hipMemcpyHostToDevice);

  ckks->evaluator.transform_to_ntt_inplace(encrypted);
  ckks->evaluator.transform_to_ntt_inplace(destination);

  hipStreamSynchronize(stream);
}

void MMEvaluator::enc_compress_ciphertext(vector<double> &values, PhantomCiphertext &ct) {
  // PhantomPlaintext zero_pt;
  // ckks->encoder.encode(0.0, ckks->scale, zero_pt);
  // PhantomCiphertext zero;
  // ckks->encryptor.encrypt(zero_pt, zero);

  auto &context_data = ckks->context->first_context_data();
  auto param = context_data.parms();

  auto coeff_modulus_size = param.coeff_modulus().size();
  auto poly_modulus_degree = ckks->degree;
  auto rns_coeff_count = poly_modulus_degree * coeff_modulus_size;

  const phantom::util::cuda_stream_wrapper &stream_wrapper = *phantom::util::global_variables::default_stream;
  const auto &stream = stream_wrapper.get_stream();

  PhantomPlaintext p;                                         // empty plaintext
  p.resize(coeff_modulus_size, poly_modulus_degree, stream);  // mod_count = 2, poly_degree = 8192 => p coeff_count = 2 * 8192 = 16384

  // // Copy vector values to GPU
  // auto val_gpu = make_cuda_auto_ptr<double>(val.size(), stream);
  // hipMemcpy(val_gpu.get(), val.data(), val.size(), hipMemcpyHostToDevice);

  // // Execute expand kernel
  auto gpu_moduli = ckks->context->gpu_rns_tables().modulus();
  // uint64_t block_size = blockDimGlb.x;
  // int num_blocks = (poly_modulus_degree + block_size - 1) / block_size;
  // expand_encode_kernel<<<num_blocks, block_size, 0, stream>>>(val_gpu.get(), poly_modulus_degree, gpu_moduli, p.data());

  hipStreamSynchronize(stream);

  auto p_data = new uint64_t[rns_coeff_count];

  // Coefficients of the two RNS polynomails should be the same except with different mod
  for (auto i = 0; i < poly_modulus_degree; i++) {
    auto coeffd = std::round(values[i] * 10000000000);
    bool is_negative = std::signbit(coeffd);
    auto coeffu = static_cast<std::uint64_t>(std::fabs(coeffd));
    if (is_negative) {
      for (std::size_t j = 0; j < 2; j++) {
        p_data[i + (j * poly_modulus_degree)] = negate_uint_mod(
            barrett_reduce_64(coeffu, param.coeff_modulus()[j]), param.coeff_modulus()[j]);
      }
    } else {
      for (std::size_t j = 0; j < 2; j++) {
        p_data[i + (j * poly_modulus_degree)] = barrett_reduce_64(coeffu, param.coeff_modulus()[j]);
      }
    }
  }

  // Debug
  cout << "Plaintext RNS polynomial coefficients:" << endl;
  for (auto i = 0; i < 10; i++) {
    cout << p_data[i] << " ";
  }
  cout << endl;

  // Copy plaintext data to GPU (synchronous)
  hipMemcpy(p.data(), p_data, rns_coeff_count * sizeof(uint64_t), hipMemcpyHostToDevice);

  // Transform all 2 RNS polynomials to the NTT domain
  nwt_2d_radix8_forward_inplace(p.data(), ckks->context->gpu_rns_tables(), coeff_modulus_size, 0, stream);

  // Debug
  hipStreamSynchronize(stream);
  hipMemcpy(p_data, p.data(), rns_coeff_count * sizeof(uint64_t), hipMemcpyDeviceToHost);
  cout << "Plaintext RNS polynomial coefficients in NTT:" << endl;
  for (auto i = 0; i < 10; i++) {
    cout << p_data[i] << " ";
  }
  cout << endl;

  // Update plaintext parameters
  p.set_chain_index(context_data.chain_index());
  p.scale() = 10000000000;

  // Encrypt zero asymmetrically
  ckks->encryptor.encrypt_zero(ct, p.chain_index());

  // (c0, c1) => (c0 + p, c1)
  uint64_t gridDimGlb = rns_coeff_count / blockDimGlb.x;
  add_rns_poly<<<gridDimGlb, blockDimGlb, 0, stream>>>(
      ct.data(), p.data(), gpu_moduli, ct.data(), poly_modulus_degree, coeff_modulus_size);

  ct.scale() = p.scale();
}

vector<PhantomCiphertext> MMEvaluator::decompress_ciphertext(const PhantomCiphertext &encrypted) {
  auto N = ckks->degree;
  uint32_t logN = ceil(log2(N));

  vector<PhantomCiphertext> temp;
  temp.push_back(encrypted);

  PhantomCiphertext tempctxt_rotated;
  PhantomCiphertext tempctxt_shifted;
  PhantomCiphertext tempctxt_rotatedshifted;

  for (uint32_t i = 0; i < logN; i++) {
    vector<PhantomCiphertext> newtemp(temp.size() * 2);
    uint32_t galois_elt = ckks->galois_elts[i];
    int index_raw = (N << 1) - (1 << i);
    int index = (index_raw * galois_elt) % (N << 1);
    for (uint32_t a = 0; a < temp.size(); a++) {
      ckks->evaluator.apply_galois(temp[a], galois_elt, *(ckks->galois_keys), tempctxt_rotated);  // sub
      ckks->evaluator.add(temp[a], tempctxt_rotated, newtemp[a]);
      multiply_power_of_x(temp[a], tempctxt_shifted, index_raw);  // x**-1
      multiply_power_of_x(tempctxt_rotated, tempctxt_rotatedshifted, index);
      ckks->evaluator.add(tempctxt_shifted, tempctxt_rotatedshifted, newtemp[a + temp.size()]);
    }
    temp = newtemp;
  }

  return temp;
}

void MMEvaluator::matrix_mul(vector<vector<double>> &x, vector<vector<double>> &y, vector<PhantomCiphertext> &res) {
  auto timer = Timer();

  vector<PhantomPlaintext> a_pts;
  a_pts.reserve(768);

  for (int i = 0; i < 768; i++) {
    PhantomPlaintext pt;
    ckks->encoder.encode(x[i], ckks->scale, pt);
    a_pts.emplace_back(pt);
  }

  timer.start();
  int b_cts_count = 768 * 64 / ckks->degree;
  vector<PhantomCiphertext> b_compressed_cts;
  b_compressed_cts.reserve(b_cts_count);

  for (int i = 0; i < b_cts_count; i++) {
    PhantomCiphertext ct;
    enc_compress_ciphertext(y[i], ct);
    b_compressed_cts.push_back(ct);
  }
  timer.stop();
  cout << "Compression took: " << timer.duration<milliseconds>() << " milliseconds" << endl;

  timer.start();
  vector<PhantomCiphertext> b_expanded_cts;
  for (auto i = 0; i < b_compressed_cts.size(); i++) {
    vector<PhantomCiphertext> temp_cts = decompress_ciphertext(b_compressed_cts[i]);
    cout << "Expanded ciphertext #" << i << endl;

    // hipStreamSynchronize(phantom::util::global_variables::default_stream->get_stream());
    // auto temp_ct0_data = new uint64_t[ckks->degree * 2];
    // hipMemcpy(temp_ct0_data, temp_cts[0].data(), ckks->degree * 2 * sizeof(uint64_t), hipMemcpyDeviceToHost);

    // for (int j = 0; j < 10; j++) {
    //   cout << temp_ct0_data[j] << " ";
    // }
    // cout << endl;

    ckks->print_decrypted_ct(temp_cts[0], 10);

    b_expanded_cts.insert(
        b_expanded_cts.end(), make_move_iterator(temp_cts.begin()), make_move_iterator(temp_cts.end()));
  }
  timer.stop();
  cout << "Decompression took: " << timer.duration<seconds>() << " seconds" << endl;

  PhantomPlaintext zero_pt;
  PhantomCiphertext zero;
  ckks->encoder.encode(0.0, ckks->scale, zero_pt);
  ckks->encryptor.encrypt(zero_pt, zero);

  timer.start();
  for (int i = 0; i < 64; i++) {
    PhantomCiphertext res_col_ct = zero;
    vector<PhantomCiphertext> temp_cts(768);
    for (int j = 0; j < 768; j++) {
      ckks->evaluator.multiply_plain(b_expanded_cts[i * 768 + j], a_pts[j], temp_cts[j]);
    }
    res_col_ct.scale() = temp_cts[0].scale();
    ckks->evaluator.add_many(temp_cts, res_col_ct);
    res_col_ct.scale() *= 4096;
    res.push_back(res_col_ct);
  }

  for (auto &ct : res) {
    while (ct.coeff_modulus_size() > 1) {
      ckks->evaluator.rescale_to_next_inplace(ct);
    }
  }
  timer.stop();
  cout << "Result calculation time: " << timer.duration<milliseconds>() << " milliseconds" << endl;
}
